#include "hip/hip_runtime.h"
/*****************************************************************************
Copyright: 2019, Yuan Yongqi.
File name: kernel.cu
Description: The parallel calculation of cpu and gpu for multi-faceted gravity
anomaly forward, the bott method for inversion of three-dimensional positive
triangle mesh interface and the nonlinear method are implemented, and the
time consumption under the forward motion of cpu and gpu is compared.
Author: Yuan Yongqi
Version: 1.3
Date: 2019.1.2
*****************************************************************************/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>
#include <assert.h>
#include <stdarg.h>

#define G 6.67e-11*1e5*1000*1000
#define PI       3.1415926
/*Relatively constant density*/
#define P      1

/*setting for GPU forward8*/
#define BLOCK_SIZE 128
/*Forward index data width*/
#define NCOLUMN 12

/*Matrix structure*/
typedef struct {
	/* DIM */
	int rows;
	int cols;
	float** data;
} Matrix;

/*Basic operation macro definition*/
#define NORM(a,b,c) sqrtf(pow((a),2) + pow((b),2) + pow((c),2))
#define DOT(a1,a2,a3,b1,b2,b3) (a1)*(b1) + (a2)*(b2) + (a3)*(b3)
#define CROSSX(a1,a2,a3,b1,b2,b3) ((a2)*(b3)-(a3)*(b2))/ \
	(sqrtf(pow((a2)*(b3) - (a3)*(b2), 2) + \
	pow(a3*b1 - a1*b3, 2) + pow(a1*b2 - b1*a2, 2)))
#define CROSSY(a1,a2,a3,b1,b2,b3) ((a3)*(b1)-(a1)*(b3))/ \
	(sqrtf(pow((a2)*(b3) - (a3)*(b2), 2) + \
	pow((a3)*(b1) - (a1)*(b3), 2) + pow((a1)*(b2) - (b1)*(a2), 2)))
#define CROSSZ(a1,a2,a3,b1,b2,b3) ((a1)*(b2)-(b1)*(a2))/ \
	(sqrtf(pow((a2)*(b3) - (a3)*(b2), 2) + \
	pow((a3)*(b1) - (a1)*(b3), 2) + pow((a1)*(b2) - (b1)*(a2), 2)))
#define SIGN(a) ((a)<0)?-1.0:(((a)<=0.000001)?0.0:1.0)

/*************************************************
Function: ReadData
Description: Read three columns of data in a file
if bFlag=1
input:const char *chFileName,
	bool bFlag, int *iNum,
	int iRow, float *aX, float *aY, float *aZ
output: The number of rows in the matrix in the file
if bFlag=0
input:const char *chFileName,
	bool bFlag, int *iNum,
	int iRow, float *aX, float *aY, float *aZ
output: Read file data is written to three arrays
*************************************************/
__host__ void ReadData(const char *chFileName,
	bool bFlag, int *iNum,
	int iRow, float *aX, float *aY, float *aZ);

/*************************************************
Function: Offset
Description : Offset the observation point(x,y)and make z=0
input : int iNum, float *aX, float *aY, float *aZ
output : float *aX, float *aY, float *aZ
*************************************************/
__host__ void Offset(int iNum, float *aX, float *aY, float *aZ);

/*************************************************
Function: MaxDiff
Description : Find the absolute value of the maximum error of two vectors
input :  float A[], float B[], int iLength
output : return
*************************************************/
__host__ float MaxDiff(float A[], float B[], int iLength);

/*************************************************
Function: getGPUInfo
Description : Get the parameter information of gpu
input : NULL
output : Print in console
*************************************************/
__host__ void getGPUInfo();

/*************************************************
Function: forwardIndex
Description : Generate triangle-specific data for
forward modeling based on tri and data
input :float *aTriX, float *aTriY, float *aTriZ, int iTriNum,
	float *aDataX, float *aDataY, float *aDataZ
output : float A[][NCOLUMN], float B[][NCOLUMN],
example:
tri:
1  4  5
point:
(datax,datay,dataz)
matrixA
point1(X,Y,Z),point4(X,Y,Z),point5(X,Y,Z),point1(X,Y,Z)
matrixB
point1(X,Y,0),point4(X,Y,0),point5(X,Y,0),point1(X,Y,0)
*************************************************/
__host__ void forwardIndex(float A[][NCOLUMN], float B[][NCOLUMN],
	float *aTriX, float *aTriY, float *aTriZ, int iTriNum,
	float *aDataX, float *aDataY, float *aDataZ);

/*************************************************
Function: lineMethod
Description :*Holstein, H., Ketteridge, B., 1996.
Gravimetric analysis of uniform polyhedra. Geophysics 61, 357�C364.
*Holstein, H., Sch��rholz, P., Starr, A. J. and Chakraborty, M., 1999,
*Comparison of gravimetric formulas for uniform
polyhedra: Geophysics, 64, 1438�C1446.
input :aObsX,aObsY,aObsZ,index[][12],star,end
output : return
*************************************************/
__host__ float lineMethod(float aObsX, float aObsY, float aObsZ,
	float index[][12], int star, int end);

/*************************************************
Function: normPrint
Description :Print the two norms of the difference between the two vectors
input :float arrA[], float arrB[], int N
output : Print in console
*************************************************/
__host__ void normPrint(float arrA[], float arrB[], int N);

/* Convert to unit matrix */
__host__ void set_identity_matrix(Matrix m);

/* Allocate initial space for the matrix */
Matrix alloc_matrix(int rows, int cols);

/* Exchange two rows of the  matrix */
__host__ void swap_rows(Matrix m, int r1, int r2);

/* Multiply a row of a matrix by a factor */
__host__ void scale_row(Matrix m, int r, float scalar);

/* Add scalar * row r2 to row r1. */
__host__ void shear_row(Matrix m, int r1, int r2, float scalar);

/* Uses Gauss-Jordan elimination.
Inversion of the matrix (learn from others)
The elimination procedure works by applying elementary row
operations to our input matrix until the input matrix is reduced to
the identity matrix.
Simultaneously, we apply the same elementary row operations to a
separate identity matrix to produce the inverse matrix.
If this makes no sense, read wikipedia on Gauss-Jordan elimination.

This is not the fastest way to invert matrices, so this is quite
possibly the bottleneck. */
__host__ int destructive_invert_matrix(Matrix input, Matrix output);

/*************************************************
Function: JacobiIndex
Description : Generate the index used to calculate the Jacobian matrix
input :float *aTriX, float *aTriY, float *aTriZ, int iTriNum,
	float *aDataX, float *aDataY, float *aDataZ,
	int maxFaceNum, int iDataNum, 
output : float A[][NCOLUMN], float B[][NCOLUMN],*aOffset
*************************************************/
__host__ void JacobiIndex(
	float *aTriX, float *aTriY, float *aTriZ, int iTriNum,
	float *aDataX, float *aDataY, float *aDataZ,
	int maxFaceNum, int iDataNum, int *aOffset);

/*************************************************
Function:JacobiCPU
Description :Cpu's Jacobian matrix calculation
input :float *aDataX, float *aDataY, float *aDataZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ,
	int iTriNum, int iDataNum, int iObsNum,
output : float *result
*************************************************/
__host__ void JacobiCPU(float *aDataX, float *aDataY, float *aDataZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ,
	int iTriNum, int iDataNum, int iObsNum,
	float *result);

/*************************************************
Function:getSolve
Description :Complete the iterative quantity based on
the Tikhonov regularization
input : int iDataNum, int iObsNum, float lambda,
	float *J, float *aObsG, float *aDataZ, 
output : float *aDataZ, 
*************************************************/
__host__ void getSolve(int iDataNum, int iObsNum, float lambda,
	float *J, float *aObsG, float *aDataZ, float *aForwardG);

/*************************************************
Function:getTopG
Description :
input :const char *chFileName,
	int iObsNum, float aObsX[], float aObsY[], float aObsZ[],
output :float *out
*************************************************/
__host__ void getTopG(const char *chFileName,
	int iObsNum, float aObsX[], float aObsY[], float aObsZ[],
	float *out);
//device function

/*************************************************
Function:  lineMethod
Description :Gpu device function
input :float obs_x, float obs_y,
float obs_z, float *index, int j
output : return
*************************************************/
__device__ float lineMethod(float obs_x, float obs_y,
	float obs_z, float *index, int j);

/*************************************************
Function:  Kernelreduce
Description :Gpu device function
input :const float *a
output : float *r
*************************************************/
__global__ void Kernelreduce(const float *a, float *r);

/*************************************************
Function:  KernelTop
Description :Gpu device function
input :float  float *d_obsX, float *d_obsY, float *d_obsZ,
	float *d_Arraytop, float *d_Arraybot,
	const int triNum, const int PointPerThreads
output : *midResult,
*************************************************/
__global__ void KernelTop(float *midResult, float *d_obsX, float *d_obsY, float *d_obsZ,
	float *d_Arraytop, 
	const int triNum, const int PointPerThreads);


//forward

__host__ void forwardGravityCPU(float *aDataX, float *aDataY, float *aDataZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ,
	int iTriNum, int iObsNum, float aTopG[],
	float *out);

void forwardGravityGPU(float *aDataX, float *aDataY, float *aDataZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ,
	int iTriNum, int iObsNum, float topG[],
	float *Forward);

//inversion


/*************************************************
Function:BottCPU
Description :Cpu bott method interface inversion
input : float *aDataX, float *aDataY, float *aObsG,
float *aTriX, float *aTriY, float *aTriZ,
int iTriNum, int iDataNum,
int iterate,
output : float *aForwardG, float *invZ
*************************************************/
__host__ void BottCPU(float *aDataX, float *aDataY, float *aObsG,
	float *aTriX, float *aTriY, float *aTriZ,
	int iTriNum, int iDataNum,
	int iterate, float *aTopG, float *aForwardG, float *invZ, bool bFlag);

/*************************************************
Function:BottGPU
Description :Gpu bott method interface inversion
input : float *aDataX, float *aDataY, float *aObsG,
float *aTriX, float *aTriY, float *aTriZ,
int iTriNum, int iDataNum,
int iterate,
output : float *aForwardG, float *invZ
*************************************************/
__host__ void BottGPU(float *aDataX, float *aDataY, float *aObsG,
	float *aTriX, float *aTriY, float *aTriZ,
	int iTriNum, int iDataNum, 
	int iterate, float *aTopG, float *aForwardG, float *invZ, bool bFlag);

/*************************************************
Function:NolineCPU
Description :Nonlinear interface inversion of cpu
input :float *aDataX, float *aDataY,
float *aTriX, float *aTriY, float *aTriZ,
float *aObsX, float *aObsY, float *aObsZ, float *aObsG,
int iTriNum, int iDataNum, int iObsNum,
int iterate, float intiValue, float lambda,float *aTopG
output :float *invZ,float *aForwardG
*************************************************/
__host__ void NolineCPU(float *aDataX, float *aDataY, float *invZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ, float *aObsG,
	int iTriNum, int iDataNum, int iObsNum,
	int iterate, float intiValue, float lambda, float *aTopG,
	float *aForwardG, bool bFlag);

/*************************************************
Function:NolineGPU
Description :Nonlinear interface inversion of gpu
input :float *aDataX, float *aDataY,
float *aTriX, float *aTriY, float *aTriZ,
float *aObsX, float *aObsY, float *aObsZ, float *aObsG,
int iTriNum, int iDataNum, int iObsNum,
int iterate, float intiValue, float lambda
output :float *invZ,float *aForwardG
*************************************************/
__host__ void NolineGPU(float *aDataX, float *aDataY, float *invZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ, float *aObsG,
	int iTriNum, int iDataNum, int iObsNum,
	int iterate, float intiValue, float lambda, float *aTopG,
	float *aForwardG, bool bFlag);


int main()
{

	//char *chDataFile = "data54_71.txt";
	//char *chTriFile = "tri54_71.txt";
	//char *chObsFile = "obs54_71.txt";
	//char *chBlnFile = "bln54_71.txt";

	//char *chDataFile = "data47_61.txt";
	//char *chTriFile = "tri47_61.txt";
	//char *chObsFile = "obs47_61.txt";
	//char *chBlnFile = "bln47_61.txt";

	//char *chDataFile = "data39_51.txt";
	//char *chTriFile = "tri39_51.txt";
	//char *chObsFile = "obs39_51.txt";
	//char *chBlnFile = "bln39_51.txt";

	//char *chDataFile = "data31_41.txt";
	//char *chTriFile = "tri31_41.txt";
	//char *chObsFile = "obs31_41.txt";
	//char *chBlnFile = "bln31_41.txt";

	//char *chDataFile = "data24_31.txt";
	//char *chTriFile = "tri24_31.txt";
	//char *chObsFile = "obs24_31.txt";
	//char *chBlnFile = "bln24_31.txt";

	//char *chDataFile = "data16_21.txt";
	//char *chTriFile = "tri16_21.txt";
	//char *chObsFile = "obs16_21.txt";
	//char *chBlnFile = "bln16_21.txt";

	//char *chDataFile = "data8_11.txt";
	//char *chTriFile = "tri8_11.txt";
	//char *chObsFile = "obs8_11.txt";
	//char *chBlnFile = "bln8_11.txt";

	char *chDataFile = "data4_6.txt";
	char *chTriFile = "tri4_6.txt";
	char *chObsFile = "obs4_6.txt";
	char *chBlnFile = "bln4_6.txt";

	int iDataNum, iTriNum, iObsNum;//ԭʼ��������//��������//��������

	ReadData(chDataFile, true, &iDataNum, 0, NULL, NULL, NULL);
	printf("iDataNum:  %d\n", iDataNum);
	float *aDataX = (float*)malloc(sizeof(float)*iDataNum);//�����ģ����ͬ
	float *aDataY = (float*)malloc(sizeof(float)*iDataNum);//�����ģ����ͬ����bott����
	float *aDataZ = (float*)malloc(sizeof(float)*iDataNum);
	ReadData(chDataFile, false, NULL, iDataNum, aDataX, aDataY, aDataZ);

	ReadData(chTriFile, true, &iTriNum, 0, NULL, NULL, NULL);
	printf("iTriNum:   %d\n", iTriNum);
	float *aTriX = (float*)malloc(sizeof(float)*iTriNum);
	float *aTriY = (float*)malloc(sizeof(float)*iTriNum);
	float *aTriZ = (float*)malloc(sizeof(float)*iTriNum);
	ReadData(chTriFile, false, NULL, iTriNum, aTriX, aTriY, aTriZ);

	ReadData(chObsFile, true, &iObsNum, 0, NULL, NULL, NULL);
	printf("iObsNum:   %d\n", iObsNum);
	float *aObsX = (float*)malloc(sizeof(float)*iObsNum);
	float *aObsY = (float*)malloc(sizeof(float)*iObsNum);
	float *aObsZ = (float*)malloc(sizeof(float)*iObsNum);
	float *aObsG = (float*)malloc(sizeof(float)*iObsNum);
	ReadData(chObsFile, false, NULL, iObsNum, aObsX, aObsY, aObsG);
	Offset(iObsNum, aObsX, aObsY, aObsZ);
	

	//getGPUInfo();

	float *aTopG = (float*)malloc(sizeof(float)*iObsNum);
	getTopG(chBlnFile, iObsNum, aObsX, aObsY, aObsZ, aTopG);

	clock_t start, finish;
	double totalTime = 0.0;

	////CPU and GPU forward
	//printf("\n     (CPU and GPU forward)\n");

	//float *aForCPU = (float*)malloc(sizeof(float)*iObsNum);
	//float *aForGPU = (float*)malloc(sizeof(float)*iObsNum);

	//printf("\n A,Cpu running\n");
	//start = clock();

	//forwardGravityCPU(aDataX, aDataY, aDataZ,
	//	aTriX, aTriY, aTriZ,
	//	aObsX, aObsY, aObsZ,
	//	iTriNum, iObsNum, aTopG,
	//	aForCPU);

	//finish = clock();
	//totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	//printf("\nThe total time is %lf seconds!\n", totalTime);

	//printf("\n B,GPU running\n");
	//start = clock();

	//forwardGravityGPU(aDataX, aDataY, aDataZ,
	//	aTriX, aTriY, aTriZ,
	//	aObsX, aObsY, aObsZ,
	//	iTriNum, iObsNum, aTopG,
	//	aForGPU);

	//finish = clock();
	//totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	//printf("\nThe total time is %lf seconds!\n", totalTime);

	//FILE *fp;
	//fp = fopen("out8_11g.txt", "w+");
	//for (int i = 0; i<iObsNum; i++)
	//	fprintf(fp, "%f  %f   %f\n ", aObsX[i],aObsY[i],aForGPU[i]);
	//fclose(fp);
	//FILE *fp1;
	//fp1 = fopen("out8_11c.txt", "w+");
	//for (int i = 0; i<iObsNum; i++)
	//	fprintf(fp1, "%f  %f   %f\n ", aObsX[i], aObsY[i], aForGPU[i]);
	//fclose(fp1);


	float lambda = 1;
	int iterate = 1;
	float intiValue = 1;
	bool show = 0;


	////Bott inversion
	//printf("\n     (Bott inversion runningtime)\n");
	//float *aInvGCPU = (float*)malloc(sizeof(float)*iObsNum);
	//float *aInvGGPU = (float*)malloc(sizeof(float)*iObsNum);
	//float *aInvZCPU = (float*)malloc(sizeof(float)*iDataNum);
	//float *aInvZGPU = (float*)malloc(sizeof(float)*iDataNum);
	//printf("\n A,CPU running\n");
	//start = clock();
	//BottCPU(aDataX, aDataY, aObsG,
	//	aTriX, aTriY, aTriZ,
	//	iTriNum, iDataNum,
	//	iterate, aTopG, aInvGCPU, aInvZCPU,show);
	//finish = clock();
	//totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	//printf("\nThe total time is %lf seconds!\n", totalTime);
	//printf("\n B,GPU running\n");
	//start = clock();
	//BottGPU(aDataX, aDataY, aObsG,
	//	aTriX, aTriY, aTriZ,
	//	iTriNum, iDataNum,
	//	iterate, aTopG, aInvGGPU, aInvZGPU, show);
	//finish = clock();
	//totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	//printf("\nThe total time is %lf seconds!\n", totalTime);




	//Nonlinear inversion
	printf("\n     (Noline inversion runningtime)\n");
	float *aNolineGCPU = (float*)malloc(sizeof(float)*iObsNum);
	float *aNolineGGPU = (float*)malloc(sizeof(float)*iObsNum);
	float *aNolineZCPU = (float*)malloc(sizeof(float)*iDataNum);
	float *aNolineZGPU = (float*)malloc(sizeof(float)*iDataNum);
	printf("\n A,Cpu running\n");
	start = clock();
	NolineCPU(aDataX, aDataY, aNolineZCPU,
		aTriX, aTriY, aTriZ,
		aObsX, aObsY, aObsZ, aObsG,
		iTriNum, iDataNum, iObsNum,
		iterate, intiValue, lambda, aTopG, aNolineGCPU, show);
	finish = clock();
	totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("\nThe total time is %lf seconds!\n", totalTime);
	printf("\n B,Gpu running\n");
	start = clock();
	NolineGPU(aDataX, aDataY, aNolineZGPU,
		aTriX, aTriY, aTriZ,
		aObsX, aObsY, aObsZ, aObsG,
		iTriNum, iDataNum, iObsNum,
		iterate, intiValue, lambda, aTopG, aNolineGGPU, show);
	finish = clock();
	totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("\nThe total time is %lf seconds!\n", totalTime);


	return 0;
}

__host__ void ReadData(const char *chFileName,
	bool bFlag, int *iNum,
	int iRow, float *aX, float *aY, float *aZ)
{
	FILE *fp;
	fp = fopen(chFileName, "r");
	if (fp == NULL)
	{
		printf("fail to open the file��\n");
	}
	if (bFlag == true)
	{
		int Count = 0;
		float Temp;
		while (fscanf(fp, "%f", &Temp) != EOF)
		{
			Count++;
		}
		*iNum = Count / 3;
	}
	else
	{
		for (int i = 0; i < iRow; i++)
		{
			fscanf(fp, "%f %f %f", &aX[i], &aY[i], &aZ[i]);
		}
	}
	fclose(fp);
}

__host__ void Offset(int iNum, float *aX, float *aY, float *aZ)
{
	for (int i = 0; i < iNum; i++)
	{
		aX[i] = aX[i] + 0.1;
		aY[i] = aY[i] + 0.1;
		aZ[i] = 0;
	}
}

__host__ float MaxDiff(float A[], float B[], int iLength)
{
	float fMaxDiff = 0;
	for (int i = 0; i < iLength; i++)
	{
		if (fMaxDiff < abs(A[i] - B[i]))
		{
			fMaxDiff = abs(A[i] - B[i]);
		}
	}
	return fMaxDiff;
}

__host__ void forwardIndex(float A[][NCOLUMN], 
	float *aTriX, float *aTriY, float *aTriZ, int iTriNum,
	float *aDataX, float *aDataY, float *aDataZ)//�½���
{
	for (int i = 0; i < iTriNum; i++)
	{
		int  a = aTriX[i] - 1;//index1to3
		int  b = aTriY[i] - 1;//index4to6
		int  c = aTriZ[i] - 1;//index7to9

		A[i][0] = aDataX[a];
		A[i][1] = aDataY[a];
		A[i][2] = aDataZ[a];

		A[i][3] = aDataX[b];
		A[i][4] = aDataY[b];
		A[i][5] = aDataZ[b];

		A[i][6] = aDataX[c];
		A[i][7] = aDataY[c];
		A[i][8] = aDataZ[c];

		A[i][9] = aDataX[a];
		A[i][10] = aDataY[a];
		A[i][11] = aDataZ[a];
	}
}

__host__ float lineMethod(float aObsX, float aObsY, float aObsZ,
	float index[][12], int star, int end)
{
	/*
	*Holstein, H., Ketteridge, B., 1996. Gravimetric analysis of uniform polyhedra. Geophysics 61, 357�C364.
	*Holstein, H., Sch��rholz, P., Starr, A. J. and Chakraborty, M., 1999,
	*Comparison of gravimetric formulas for uniform polyhedra: Geophysics, 64, 1438�C1446.
	*input:obs(x,y,z),index,row of index
	*output:value for the uniform polyherdra
	*/
	float CalcGravity = 0;
	for (int j = star; j < end; j++)
	{
		float a1 = index[j][3] - index[j][0],
			a2 = index[j][4] - index[j][1],
			a3 = index[j][5] - index[j][2];
		float b1 = index[j][6] - index[j][3],
			b2 = index[j][7] - index[j][4],
			b3 = index[j][8] - index[j][5];

		float dnx = CROSSX(a1, a2, a3, b1, b2, b3);
		float dny = CROSSY(a1, a2, a3, b1, b2, b3);
		float dnz = CROSSZ(a1, a2, a3, b1, b2, b3);

		float lineGravity = 0;
		for (int i = 0; i < 3; i++)
		{
			float a1 = index[j][i * 3],
				a2 = index[j][i * 3 + 1],
				a3 = index[j][i * 3 + 2];
			float b1 = index[j][i * 3 + 3],
				b2 = index[j][i * 3 + 4],
				b3 = index[j][i * 3 + 5];

			float L = NORM(a1 - b1, a2 - b2, a3 - b3);

			float dtx = (b1 - a1) / L;
			float dty = (b2 - a2) / L;
			float dtz = (b3 - a3) / L;

			float dhx = CROSSX(dtx, dty, dtz, dnx, dny, dnz);
			float dhy = CROSSY(dtx, dty, dtz, dnx, dny, dnz);
			float dhz = CROSSZ(dtx, dty, dtz, dnx, dny, dnz);

			float vr1x = a1 - aObsX;
			float vr1y = a2 - aObsY;
			float vr1z = a3 - aObsZ;

			float vr2x = b1 - aObsX;
			float vr2y = b2 - aObsY;
			float vr2z = b3 - aObsZ;

			float v = DOT(dnx, dny, dnz, vr1x, vr1y, vr1z);
			float h = DOT(dhx, dhy, dhz, vr2x, vr2y, vr2z);
			float r1 = NORM(vr1x, vr1y, vr1z);
			float r2 = NORM(vr2x, vr2y, vr2z);

			float DN = L / (r2 + r1);

			float C = 2 * atanh(DN);
			float E = 0.5*(r1 + r2 - L*DN);
			float Q = (SIGN(v)) * 2 * atan(h*DN / (abs(v) + E));

			lineGravity = lineGravity + dnz*(h*C - v*Q);
		}
		CalcGravity = CalcGravity + lineGravity;
	}
	return CalcGravity;
}

__device__ float lineMethod(float obs_x, float obs_y,
	float obs_z, float* index, int j)
{
	float a1 = index[j * 12 + 3] - index[j * 12 + 0],
		a2 = index[j * 12 + 4] - index[j * 12 + 1],
		a3 = index[j * 12 + 5] - index[j * 12 + 2];
	float b1 = index[j * 12 + 6] - index[j * 12 + 3],
		b2 = index[j * 12 + 7] - index[j * 12 + 4],
		b3 = index[j * 12 + 8] - index[j * 12 + 5];

	float dnx = CROSSX((double)a1, (double)a2, (double)a3, 
		(double)b1, (double)b2, (double)b3);
	float dny = CROSSY((double)a1, (double)a2, (double)a3,
		(double)b1, (double)b2, (double)b3);
	float dnz = CROSSZ((double)a1, (double)a2, (double)a3,
		(double)b1, (double)b2, (double)b3);

	float LineGravity = 0;
	for (int i = 0; i < 3; i++)//�߻���
	{
		float a1 = index[j * 12 + i * 3], a2 = index[j * 12 + i * 3 + 1], a3 = index[j * 12 + i * 3 + 2];
		float b1 = index[j * 12 + i * 3 + 3], b2 = index[j * 12 + i * 3 + 4], b3 = index[j * 12 + i * 3 + 5];


		float L = NORM((double)a1 - (double)b1, (double)a2 - (double)b2, (double)a3 - (double)b3);

		//printf("L: %f\n", L);

		float dtx = ((double)b1 - (double)a1) / (double)L;
		float dty = ((double)b2 - (double)a2) / (double)L;
		float dtz = ((double)b3 - (double)a3) / (double)L;

		float dhx = CROSSX((double)dtx, (double)dty, (double)dtz, 
			(double)dnx, (double)dny, (double)dnz);
		float dhy = CROSSY((double)dtx, (double)dty, (double)dtz,
			(double)dnx, (double)dny, (double)dnz);
		float dhz = CROSSZ((double)dtx, (double)dty, (double)dtz,
			(double)dnx, (double)dny, (double)dnz);

		float vr1x = (double)a1 - (double)obs_x;
		float vr1y = (double)a2 - (double)obs_y;
		float vr1z = (double)a3 - (double)obs_z;

		float vr2x = (double)b1 - (double)obs_x;
		float vr2y = (double)b2 - (double)obs_y;
		float vr2z = (double)b3 - (double)obs_z;

		float v = DOT((double)dnx, (double)dny, (double)dnz,
			(double)vr1x, (double)vr1y, (double)vr1z);
		float h = DOT((double)dhx, (double)dhy, (double)dhz,
			(double)vr2x, (double)vr2y, (double)vr2z);
		float r1 = NORM((double)vr1x, (double)vr1y, (double)vr1z);
		float r2 = NORM((double)vr2x, (double)vr2y, (double)vr2z);


		float DN = (double)L / ((double)r2 + (double)r1);
		//printf("DN: %f\n", DN);
		float C = (double)2 * (double)atanh((double)DN);
		//printf("C: %f\n",C);

		float E = (double)0.5*((double)r1 + (double)r2 - (double)L*(double)DN);
		float Q = (double)(SIGN(v)) * (double)2.0 * atan((double)h*(double)DN / (abs((double)v) + (double)E));

		LineGravity = LineGravity + (double)dnz*((double)h*(double)C - (double)v*(double)Q);
		//printf("LineGravity: %f\n", LineGravity);
	}

	return LineGravity;
}


__host__ void normPrint(float arrA[], float arrB[], int N)
{
	float sumNorm = 0;
	for (int i = 0; i < N; i++)
	{
		sumNorm += pow((arrA[i] - arrB[i]), 2);
	}
	printf("\n norm:   %f", sqrt(sumNorm));
}

__host__ void set_identity_matrix(Matrix m) {
	int i;
	int j;
	assert(m.rows == m.cols);
	for (i = 0; i < m.rows; ++i) {
		for (j = 0; j < m.cols; ++j) {
			if (i == j) {
				m.data[i][j] = 1.0;
			}
			else {
				m.data[i][j] = 0.0;
			}
		}
	}
}

Matrix alloc_matrix(int rows, int cols) {
	Matrix m;
	int i;
	int j;
	m.rows = rows;
	m.cols = cols;
	m.data = (float**)malloc(sizeof(float*)* m.rows);

	for (i = 0; i < m.rows; ++i)
	{
		m.data[i] = (float*)malloc(sizeof(float)* m.cols);
		assert(m.data[i]);
		for (j = 0; j < m.cols; ++j) {
			m.data[i][j] = 0.0;
		}
	}
	return m;
}

__host__ void swap_rows(Matrix m, int r1, int r2) {
	float *tmp;
	assert(r1 != r2);
	tmp = m.data[r1];
	m.data[r1] = m.data[r2];
	m.data[r2] = tmp;
}

__host__ void scale_row(Matrix m, int r, float scalar) {
	int i;
	assert(scalar != 0.0);
	for (i = 0; i < m.cols; ++i) {
		m.data[r][i] *= scalar;
	}
}

__host__ void shear_row(Matrix m, int r1, int r2, float scalar) {
	int i;
	assert(r1 != r2);
	for (i = 0; i < m.cols; ++i) {
		m.data[r1][i] += scalar * m.data[r2][i];
	}
}

__host__ int destructive_invert_matrix(Matrix input, Matrix output) {
	int i;
	int j;
	int r;
	float scalar;
	float shear_needed;
	assert(input.rows == input.cols);
	assert(input.rows == output.rows);
	assert(input.rows == output.cols);

	set_identity_matrix(output);

	/* Convert input to the identity matrix via elementary row operations.
	The ith pass through this loop turns the element at i,i to a 1
	and turns all other elements in column i to a 0. */

	for (i = 0; i < input.rows; ++i) {

		if (input.data[i][i] == 0.0) {
			/* We must swap rows to get a nonzero diagonal element. */

			for (r = i + 1; r < input.rows; ++r) {
				if (input.data[r][i] != 0.0) {
					break;
				}
			}
			if (r == input.rows) {
				/* Every remaining element in this column is zero, so this
				matrix cannot be inverted. */
				return 0;
			}
			swap_rows(input, i, r);
			swap_rows(output, i, r);
		}

		/* Scale this row to ensure a 1 along the diagonal.
		We might need to worry about overflow from a huge scalar here. */
		scalar = 1.0 / input.data[i][i];
		scale_row(input, i, scalar);
		scale_row(output, i, scalar);

		/* Zero out the other elements in this column. */
		for (j = 0; j < input.rows; ++j) {
			if (i == j) {
				continue;
			}
			shear_needed = -input.data[j][i];
			shear_row(input, j, i, shear_needed);
			shear_row(output, j, i, shear_needed);
		}
	}

	return 1;
}

__host__ void JacobiIndex(float A[][NCOLUMN], float B[][NCOLUMN],
	float *aTriX, float *aTriY, float *aTriZ, int iTriNum,
	float *aDataX, float *aDataY, float *aDataZ, int maxFaceNum, int iDataNum, int *aOffset)//,int *end
{
	float *flagA = (float*)malloc(sizeof(float)*iDataNum*iTriNum);
	int *J = (int*)malloc(sizeof(int)*maxFaceNum*iDataNum);
	for (int i = 0; i < iDataNum; i++)
	{
		for (int j = 0; j < iTriNum; j++)
		{
			flagA[i*iTriNum + j] = 0;
			if (aTriX[j] - 1 == i)
				flagA[i*iTriNum + j] = 1;
			if (aTriY[j] - 1 == i)
				flagA[i*iTriNum + j] = 1;
			if (aTriZ[j] - 1 == i)
				flagA[i*iTriNum + j] = 1;
		}
	}

	for (int i = 0; i < iDataNum; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			J[i*maxFaceNum + j] = -1;
		}
	}

	for (int i = 0; i < iDataNum; i++)
	{
		int k = 0;
		for (int j = 0; j < iTriNum; j++)
		{
			if (flagA[i*iTriNum + j] == 1)
			{
				J[i*maxFaceNum + k] = j;
				//printf("%d\t", J[i*maxFaceNum + k]);
				k++;
			}
		}
		//array[i] = k;
		//printf("\n");
	}

	for (int k = 0; k < iDataNum; k++)
	{
		for (int i = 0; i < maxFaceNum; i++)
		{
			for (int j = 0; j < 12; j++)
			{
				A[i + k*maxFaceNum][j] = 0;
				B[i + k*maxFaceNum][j] = 0;
			}
		}
	}

	for (int n = 0; n < iDataNum; n++)
	{
		int face = 0;
		for (int i = 0; i < maxFaceNum; i++)
		{
			int k = J[i + n*maxFaceNum];
			if (k >= 0)
			{
				face++;
				int  a = aTriX[k] - 1;//index1to3
				int  b = aTriY[k] - 1;//index4to6
				int  c = aTriZ[k] - 1;//index7to9

				B[i + n*maxFaceNum][0] = aDataX[a];
				B[i + n*maxFaceNum][1] = aDataY[a];
				B[i + n*maxFaceNum][2] = aDataZ[a];
				B[i + n*maxFaceNum][3] = aDataX[b];
				B[i + n*maxFaceNum][4] = aDataY[b];
				B[i + n*maxFaceNum][5] = aDataZ[b];
				B[i + n*maxFaceNum][6] = aDataX[c];
				B[i + n*maxFaceNum][7] = aDataY[c];
				B[i + n*maxFaceNum][8] = aDataZ[c];
				B[i + n*maxFaceNum][9] = aDataX[a];
				B[i + n*maxFaceNum][10] = aDataY[a];
				B[i + n*maxFaceNum][11] = aDataZ[a];

				aDataZ[n] = aDataZ[n] + 1;//����

				A[i + n*maxFaceNum][0] = aDataX[a];
				A[i + n*maxFaceNum][1] = aDataY[a];
				A[i + n*maxFaceNum][2] = aDataZ[a];
				A[i + n*maxFaceNum][3] = aDataX[b];
				A[i + n*maxFaceNum][4] = aDataY[b];
				A[i + n*maxFaceNum][5] = aDataZ[b];//����
				A[i + n*maxFaceNum][6] = aDataX[c];
				A[i + n*maxFaceNum][7] = aDataY[c];
				A[i + n*maxFaceNum][8] = aDataZ[c];
				A[i + n*maxFaceNum][9] = aDataX[a];
				A[i + n*maxFaceNum][10] = aDataY[a];
				A[i + n*maxFaceNum][11] = aDataZ[a];

				aDataZ[n] = aDataZ[n] - 1;//�ָ�
			}
			aOffset[n] = face;
		}
	}
	free(flagA);
	free(J);
}

__host__ void JacobiCPU(float *aDataX, float *aDataY, float *aDataZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ,
	int iTriNum, int iDataNum, int iObsNum,
	float *result)
{
	int maxFaceNum = 6;
	float(*h_jbot)[NCOLUMN] = (float(*)[NCOLUMN])
		malloc(sizeof(float)* NCOLUMN * maxFaceNum*iDataNum);//�����ڴ�ռ�//Gpu
	float(*h_jtop)[NCOLUMN] = (float(*)[NCOLUMN])
		malloc(sizeof(float)* NCOLUMN * maxFaceNum*iDataNum);//row=iDataNum*maxFaceNum //Gpu
	int *offset = (int*)malloc(sizeof(int)*iDataNum);//Gpu

	JacobiIndex(h_jtop, h_jbot, aTriX, aTriY, aTriZ, iTriNum,
		aDataX, aDataY, aDataZ, maxFaceNum, iDataNum, offset);

	int star = 0;
	for (int i = 0; i < iDataNum; i++)
	{
		star = i* maxFaceNum;
		for (int j = 0; j < iObsNum; j++)
		{
			result[j*iDataNum + i] =
				-G*P*(lineMethod(aObsX[j], aObsY[j], aObsZ[j], h_jtop, star, star + offset[i])
				- lineMethod(aObsX[j], aObsY[j], aObsZ[j], h_jbot, star, star + offset[i]));
			//printf("%f\t", Jaco[j*iDataNum + i]);	
		}
		//printf("\n");
	}

	free(h_jbot);
	free(h_jtop);
	free(offset);
}

__host__ void getSolve(int iDataNum, int iObsNum, float lambda,
	float*J, float* aObsG, float* aDataZ, float* aForwardG)
{
	float(*Jt) = (float*)malloc(sizeof(float)*iObsNum*iDataNum);
	float *Jtd = (float*)malloc(sizeof(float)*iDataNum);
	float *d = (float*)malloc(sizeof(float)*iObsNum);
	float *dp = (float*)malloc(sizeof(float)*iDataNum);
	float **H = (float**)malloc(sizeof(float*)*iDataNum);
	for (int i = 0; i < iDataNum; i++)
		H[i] = (float*)malloc(sizeof(float)*iDataNum);

	for (int i = 0; i < iObsNum; i++)
		d[i] = aObsG[i] - aForwardG[i];

	//for (int i = 0; i < iObsNum; i++)
	//{
	//	printf("%f\n",d[i]);
	//}
	//for (int i = 0; i < iDataNum; i++)
	//{
	//	for (int j = 0; j < iObsNum; j++)
	//	{
	//		printf("%f\t", J[j*iDataNum + i]);
	//	}
	//	printf("\n");
	//}

	for (int i = 0; i < iDataNum; i++)
	{
		for (int j = 0; j < iObsNum; j++)
		{
			Jt[j + i*iObsNum] = J[j*iDataNum + i];
		}
	}

	for (int i = 0; i < iDataNum; i++)
	{
		for (int j = 0; j < iDataNum; j++)
		{
			H[i][j] = 0.0;
		}
	}

	for (int i = 0; i < iDataNum; i++)
	{
		for (int j = 0; j < iDataNum; j++)
		{
			for (int k = 0; k < iObsNum; k++)
			{
				H[i][j] = H[i][j] + Jt[i*iObsNum + k] * J[k*iDataNum + j];
			}
		}
	}

	for (int j = 0; j < iDataNum; j++)
		H[j][j] += lambda;//inv
	//for (int i = 0; i < iDataNum; i++)
	//{
	//	for (int j = 0; j < iDataNum; j++)
	//	{
	//		printf("%f", H[i][j]);
	//	}
	//}

	for (int i = 0; i < iDataNum; i++)
		Jtd[i] = 0;

	for (int i = 0; i < iDataNum; i++)
	{
		for (int j = 0; j < iObsNum; j++)
		{
			Jtd[i] = Jtd[i] + Jt[i*iObsNum + j] * d[j];
		}
	}

	Matrix H_mat = { iDataNum, iDataNum, H };
	Matrix inv_H_mat = alloc_matrix(iDataNum, iDataNum);

	int a = destructive_invert_matrix(H_mat, inv_H_mat);
	if (a != 1)
	{
		printf("Matrix det =0");
	}

	for (int i = 0; i < iDataNum; i++)
	{
		dp[i] = 0;
	}

	for (int i = 0; i < iDataNum; i++)
	{
		for (int j = 0; j < inv_H_mat.cols; j++)
		{
			dp[i] = dp[i] + inv_H_mat.data[i][j] * Jtd[j];
		}
	}

	for (int i = 0; i < iDataNum; i++)
	{
		aDataZ[i] = aDataZ[i] + dp[i];
		//printf("%f\n", aDataZ[i]);
	}
}

__host__ void getGPUInfo()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int dev;
	for (dev = 0; dev < deviceCount; dev++)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		if (dev == 0)
		{
			if (/*deviceProp.major==9999 && */deviceProp.minor = 9999 && deviceProp.major == 9999)
				printf("\n");

		}
		printf("\nDevice%d:\"%s\"\n", dev, deviceProp.name);
		printf("Total amount of global memory                   %u bytes\n",
			deviceProp.totalGlobalMem);
		printf("Number of mltiprocessors                        %d\n",
			deviceProp.multiProcessorCount);
		printf("Total amount of constant memory:                %u bytes\n",
			deviceProp.totalConstMem);
		printf("Total amount of shared memory per block         %u bytes\n",
			deviceProp.sharedMemPerBlock);
		printf("Total number of registers available per block:  %d\n",
			deviceProp.regsPerBlock);
		printf("Warp size                                       %d\n",
			deviceProp.warpSize);
		printf("Maximum number of threada per block:            %d\n",
			deviceProp.maxThreadsPerBlock);
		printf("Maximum sizes of each dimension of a block:     %d x %d x %d\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf("Maximum size of each dimension of a grid:       %d x %d x %d\n",
			deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
		printf("Maximum memory pitch :                          %u bytes\n",
			deviceProp.memPitch);
		printf("Texture alignmemt                               %u bytes\n",
			deviceProp.texturePitchAlignment);
		printf("Clock rate                                      %.2f GHz\n",
			deviceProp.clockRate*1e-6f);
	}
	printf("\nTest PASSED\n");
}

__host__ void getTopG(const char *chFileName,
	int iObsNum, float aObsX[], float aObsY[], float aObsZ[],
	float *out)
{
	FILE *fp;
	fp = fopen(chFileName, "r");
	if (fp == NULL)
	{
		printf("fail to open the file��\n");
	}
	int iBlnNum = 0;
	float Temp;
	while (fscanf(fp, "%f", &Temp) != EOF)
	{
		iBlnNum++;
	}
	iBlnNum = iBlnNum / 2;
	//printf("%d\n",iBlnNum);
	fclose(fp);

	fp = fopen(chFileName, "r");

	float *aBlnX = (float*)malloc(sizeof(float)*iBlnNum);
	float *aBlnY = (float*)malloc(sizeof(float)*iBlnNum);

	for (int i = 0; i < iBlnNum; i++)
	{
		fscanf(fp, "%f   %f", &aBlnX[i], &aBlnY[i]);
	}
	fclose(fp);

	/*for (int i = 0; i < iBlnNum; i++)
	{
	printf("%f\t  %f\n",aBlnX[i],aBlnY[i]);
	}*/

	float dnx = 0;
	float dny = 0;
	float dnz = 1.0;


	for (int j = 0; j < iObsNum; j++)
	{

		//float a1 = index[j][3] - index[j][0],
		//	a2 = index[j][4] - index[j][1],
		//	a3 = index[j][5] - index[j][2];
		//float b1 = index[j][6] - index[j][3],
		//	b2 = index[j][7] - index[j][4],
		//	b3 = index[j][8] - index[j][5];

		//float dnx = CROSSX(a1, a2, a3, b1, b2, b3);
		//float dny = CROSSY(a1, a2, a3, b1, b2, b3);
		//float dnz = CROSSZ(a1, a2, a3, b1, b2, b3);

		float lineGravity = 0;
		for (int i = 0; i < iBlnNum - 1; i++)
		{

			float a1 = aBlnX[i],
				a2 = aBlnY[i],
				a3 = 0;
			float b1 = aBlnX[i + 1],
				b2 = aBlnY[i + 1],
				b3 = 0;

			float L = NORM(a1 - b1, a2 - b2, a3 - b3);

			float dtx = (b1 - a1) / L;
			float dty = (b2 - a2) / L;
			float dtz = (b3 - a3) / L;

			float dhx = CROSSX(dtx, dty, dtz, dnx, dny, dnz);
			float dhy = CROSSY(dtx, dty, dtz, dnx, dny, dnz);
			float dhz = CROSSZ(dtx, dty, dtz, dnx, dny, dnz);

			float vr1x = a1 - aObsX[j];
			float vr1y = a2 - aObsY[j];
			float vr1z = a3 - aObsZ[j];

			float vr2x = b1 - aObsX[j];
			float vr2y = b2 - aObsY[j];
			float vr2z = b3 - aObsZ[j];

			float v = DOT(dnx, dny, dnz, vr1x, vr1y, vr1z);
			float h = DOT(dhx, dhy, dhz, vr2x, vr2y, vr2z);
			float r1 = NORM(vr1x, vr1y, vr1z);
			float r2 = NORM(vr2x, vr2y, vr2z);

			float DN = L / (r2 + r1);

			float C = 2 * atanh(DN);
			float E = 0.5*(r1 + r2 - L*DN);
			float Q = (SIGN(v)) * 2 * atan(h*DN / (abs(v) + E));

			lineGravity = lineGravity + dnz*(h*C - v*Q);
		}
		out[j] = lineGravity;
	}



	free(aBlnX);
	free(aBlnY);
}

__host__ void BottCPU(float *aDataX, float *aDataY, float *aObsG,
	float *aTriX, float *aTriY, float *aTriZ,
	int iTriNum, int iDataNum,
	int iterate, float *aTopG,
	float *aForwardG, float *invZ, bool bFlag)
{
	int iObsNum = iDataNum;
	float *aObsX = (float*)malloc(sizeof(float)*iObsNum);
	float *aObsY = (float*)malloc(sizeof(float)*iObsNum);
	float *aObsZ = (float*)malloc(sizeof(float)*iObsNum);
	for (int i = 0; i < iDataNum; i++)
	{
		aObsX[i] = aDataX[i] + 0.001;
		aObsY[i] = aDataY[i] + 0.001;
		aObsZ[i] = 0;
	}
	float a = 0;
	for (int i = 0; i < iDataNum; i++)
	{
		invZ[i] = (aObsG[i] * P) / (41.98*P*P + a*aObsG[i]);
	}
	for (int i = 0; i < iterate; i++)
	{
		forwardGravityCPU(aDataX, aDataY, invZ,
			aTriX, aTriY, aTriZ,
			aObsX, aObsY, aObsZ,
			iTriNum, iObsNum, aTopG,
			aForwardG);

		for (int i = 0; i < iDataNum; i++)
		{
			invZ[i] = invZ[i] + (aObsG[i] - aForwardG[i]) / (2 * PI*P*G);
		}
		if (bFlag == true)
		{
			normPrint(aObsG, aForwardG, iObsNum);
		}
	}
	free(aObsX);
	free(aObsY);
	free(aObsZ);
}

__host__ void BottGPU(float *aDataX, float *aDataY, float *aObsG,
	float *aTriX, float *aTriY, float *aTriZ,
	int iTriNum, int iDataNum,
	int iterate, float *aTopG,
	float *aForwardG, float *invZ, bool bFlag)
{
	int iObsNum = iDataNum;
	float *aObsX = (float*)malloc(sizeof(float)*iObsNum);
	float *aObsY = (float*)malloc(sizeof(float)*iObsNum);
	float *aObsZ = (float*)malloc(sizeof(float)*iObsNum);
	for (int i = 0; i < iDataNum; i++)
	{
		aObsX[i] = aDataX[i] + 0.001;
		aObsY[i] = aDataY[i] + 0.001;
		aObsZ[i] = 0;
	}
	float a = 0;
	for (int i = 0; i < iDataNum; i++)
	{
		invZ[i] = (aObsG[i] * P) / (41.98*P*P + a*aObsG[i]);
	}
	for (int i = 0; i < iterate; i++)
	{
		forwardGravityGPU(aDataX, aDataY, invZ,
			aTriX, aTriY, aTriZ,
			aObsX, aObsY, aObsZ,
			iTriNum, iObsNum, aTopG,
			aForwardG);

		for (int i = 0; i < iDataNum; i++)
		{
			invZ[i] = invZ[i] + (aObsG[i] - aForwardG[i]) / (2 * PI*P*G);
		}
		if (bFlag == true)
		{
			normPrint(aObsG, aForwardG, iObsNum);
		}
	}
	free(aObsX);
	free(aObsY);
	free(aObsZ);
}

__host__ void NolineCPU(float *aDataX, float *aDataY, float *invZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ, float *aObsG,
	int iTriNum, int iDataNum, int iObsNum,
	int iterate, float intiValue, float lambda, float *aTopG,
	float *aForwardG, bool bFlag)
{
	float *J = (float*)malloc(sizeof(float)*iObsNum*iDataNum);
	for (int i = 0; i < iDataNum; i++)
	{
		invZ[i] = intiValue;
	}
	for (int i = 0; i < iterate; i++)
	{
		forwardGravityCPU(aDataX, aDataY, invZ,
			aTriX, aTriY, aTriZ,
			aObsX, aObsY, aObsZ,
			iTriNum, iObsNum, aTopG,
			aForwardG);
		JacobiCPU(aDataX, aDataY, invZ,
			aTriX, aTriY, aTriZ,
			aObsX, aObsY, aObsZ,
			iTriNum, iDataNum, iObsNum,
			J);
		getSolve(iDataNum, iObsNum, lambda,
			J, aObsG, invZ, aForwardG);
		if (bFlag == true)
		{
			normPrint(aObsG, aForwardG, iObsNum);
		}
	}
	free(J);
}

__host__ void NolineGPU(float *aDataX, float *aDataY, float *invZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ, float *aObsG,
	int iTriNum, int iDataNum, int iObsNum,
	int iterate, float intiValue, float lambda, float *aTopG,
	float *aForwardG,bool bFlag)
{
	float *J = (float*)malloc(sizeof(float)*iObsNum*iDataNum);
	for (int i = 0; i < iDataNum; i++)
	{
		invZ[i] = intiValue;
	}
	for (int i = 0; i < iterate; i++)
	{
		forwardGravityGPU(aDataX, aDataY, invZ,
			aTriX, aTriY, aTriZ,
			aObsX, aObsY, aObsZ,
			iTriNum, iObsNum, aTopG,
			aForwardG);
		JacobiCPU(aDataX, aDataY, invZ,
			aTriX, aTriY, aTriZ,
			aObsX, aObsY, aObsZ,
			iTriNum, iDataNum, iObsNum,
			J);
		getSolve(iDataNum, iObsNum, lambda,
			J, aObsG, invZ, aForwardG);
		if (bFlag == true)
		{
			normPrint(aObsG, aForwardG, iObsNum);
		}
	}
	free(J);
}

__global__ void Kernelreduce(const float *a, float *r)
{
	__shared__ float cache[BLOCK_SIZE];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int cacheIndex = threadIdx.x;

	// copy data to shared memory from global memory
	cache[cacheIndex] = a[tid];
	__syncthreads();

	// add these data using reduce
	for (int i = blockDim.x / 2; i > 0; i /= 2)
	{
		if (cacheIndex < i)
		{
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
	}

	// copy the result of reduce to global memory
	if (cacheIndex == 0)
		r[blockIdx.x] = cache[cacheIndex];
}

__global__ void KernelTop(float *midResult, float *d_obsX, float *d_obsY, float *d_obsZ,
	float *d_Arraytop,
	const int triNum, const int PointPerThreads)
{
	//__shared__ int sdata[128];
	int idx = (blockIdx.x*blockDim.x) + threadIdx.x;//iTriNum
	int obsIndex = idx / PointPerThreads;
	int triIndex = idx % PointPerThreads;
	//int cacheIndex = threadIdx.x;

	if (triIndex<triNum)
	{
		midResult[idx] = (double)lineMethod(d_obsX[obsIndex], d_obsY[obsIndex], d_obsZ[obsIndex], d_Arraytop, triIndex);
	}
}

void forwardGravityGPU(float *aDataX, float *aDataY, float *aDataZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ,
	int iTriNum, int iObsNum, float topG[],
	float *Forward)
{
	int PointPerblock = (iTriNum%BLOCK_SIZE>0) ? (iTriNum / BLOCK_SIZE + 1) : (iTriNum / BLOCK_SIZE);
	int blockNum = iObsNum*PointPerblock;
	dim3 dimgrid(blockNum), dimblock(BLOCK_SIZE);


	int PointPerThreads = PointPerblock*BLOCK_SIZE;
	int midResultNum = blockNum*BLOCK_SIZE;
	int resultNum = iObsNum*PointPerblock;

	//printf("PointPerblock:  %d\n", PointPerblock);
	//printf("test: %d  %d\n", iTriNum, PointPerblock*BLOCK_SIZE);
	//printf("trinum*obsnum:  %d\n", iTriNum*iObsNum);
	//printf("blockNum*BLOCK_SIZE:  %d\n", blockNum*BLOCK_SIZE);
	//printf("blockNum: %d \n", blockNum);
	//printf("resultNum: %d \n", resultNum);

	const int result_bytes = sizeof(float)*resultNum;
	const int Array_bytes = sizeof(float)*iObsNum;
	const int midResult_bytes = sizeof(float)*midResultNum;
	const int FaceIndex_bytes = 12 * iTriNum * sizeof(float);

	float *result = (float*)malloc(result_bytes);
	//float *midResult = (float*)malloc(midResult_bytes);

	float(*h_indexTop)[NCOLUMN] = (float(*)[NCOLUMN])malloc(sizeof(float)* NCOLUMN * iTriNum);
	//float(*h_indexBot)[NCOLUMN] = (float(*)[NCOLUMN])malloc(sizeof(float)* NCOLUMN * iTriNum);

	forwardIndex(h_indexTop,aTriX, aTriY, aTriZ, iTriNum, aDataX, aDataY, aDataZ);

	float* d_obsX;
	float* d_obsY;
	float* d_obsZ;
	float* d_Arraytop;
	//float* d_Arraybot;

	float* d_midResult;
	float* d_result;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	cudaStatus = hipMalloc((void**)&d_obsX, Array_bytes);
	cudaStatus = hipMalloc((void**)&d_obsY, Array_bytes);
	cudaStatus = hipMalloc((void**)&d_obsZ, Array_bytes);
	cudaStatus = hipMalloc((void**)&d_Arraytop, FaceIndex_bytes);
	//cudaStatus = hipMalloc((void**)&d_Arraybot, FaceIndex_bytes);

	cudaStatus = hipMemcpy(d_obsX, aObsX, Array_bytes, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_obsY, aObsY, Array_bytes, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_obsZ, aObsZ, Array_bytes, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_Arraytop, h_indexTop, FaceIndex_bytes, hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(d_Arraybot, h_indexBot, FaceIndex_bytes, hipMemcpyHostToDevice);

	cudaStatus = hipMalloc((void**)&d_midResult, midResult_bytes);
	cudaStatus = hipMalloc((void**)&d_result, result_bytes);

	KernelTop << < dimgrid, dimblock >> >
		(d_midResult, d_obsX, d_obsY, d_obsZ, d_Arraytop,
		iTriNum, PointPerThreads);
	Kernelreduce << < dimgrid, dimblock >> >
		(d_midResult, d_result);


	cudaStatus = hipMemcpy(result, d_result, result_bytes, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	for (int i = 0; i < iObsNum; i++)
	{
		float sum = 0;
		for (int j = 0; j < PointPerblock; j++)
		{
			sum = sum + result[j + i*PointPerblock];
		}
		Forward[i] = -(sum + topG[i])*P*G;
	}

	//free(h_indexTop);
	hipFree(d_Arraytop);
	hipFree(d_obsX);
	hipFree(d_obsY);
	hipFree(d_obsZ);
	hipFree(d_midResult);
	hipFree(d_result);

}

__host__ void forwardGravityCPU(float *aDataX, float *aDataY, float *aDataZ,
	float *aTriX, float *aTriY, float *aTriZ,
	float *aObsX, float *aObsY, float *aObsZ,
	int iTriNum, int iObsNum, float aTopG[],
	float *out)
{
	const int iRows = iTriNum;
	float(*h_indexTop)[NCOLUMN] = (float(*)[NCOLUMN])malloc(sizeof(float)* NCOLUMN * iRows);
	//float(*h_indexBot)[NCOLUMN] = (float(*)[NCOLUMN])malloc(sizeof(float)* NCOLUMN * iRows);
	forwardIndex(h_indexTop, aTriX, aTriY, aTriZ, iTriNum, aDataX, aDataY, aDataZ);

	for (int i = 0; i < iObsNum; i++)
	{
		//out[i] = lineMethod(aObsX[i], aObsY[i], aObsZ[i], h_indexBot, 0, iTriNum);
		out[i] = -(lineMethod(aObsX[i], aObsY[i], aObsZ[i], h_indexTop, 0, iTriNum) + aTopG[i])*P*G;
	}

	//free(h_indexBot);
	free(h_indexTop);
}